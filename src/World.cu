#include "hip/hip_runtime.h"
#include <World.h>

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;

    Point *p = new Point(1, 2);
    c[i] = p->getX() + p->getY() - 3 + a[i] + b[i];

    delete p;
}

__global__ void propagateParticlesKernel(Particle *particles)
{
    int i = threadIdx.x;
    particles[i].x = 1;
    particles[i].y = 2;
    particles[i].z = 3;
    particles[i].vx = 4;
    particles[i].vy = 5;
    particles[i].vz = 6;
}

Data::Data(unsigned int size)
{
    this->size = size;
    fprintf(stderr, "Data Constructor!!!\n");

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void **)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMalloc((void **)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMalloc((void **)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
    }

    cudaStatus = hipMalloc((void **)&dev_parts, size * sizeof(Particle));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
    }
}

Data::~Data()
{
    fprintf(stderr, "Data Destructor!!!\n");
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_parts);
}

void Data::copyFromHostToDevice(const int *host_data, int *dev_data)
{
    cudaStatus = hipMemcpy(dev_data, host_data, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
    }
}

void Data::copyFromDeviceToHost(const int *dev_data, int *host_data)
{
    cudaStatus = hipMemcpy(host_data, dev_data, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
    }
}

void Data::copyFromHostToDevice(const Particle *host_part)
{
    cudaStatus = hipMemcpy(dev_parts, host_part, size * sizeof(Particle), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
    }
}

void Data::copyFromDeviceToHost(Particle *host_part)
{
    cudaStatus = hipMemcpy(host_part, dev_parts, size * sizeof(Particle), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
    }
}

Summator::Summator(std::shared_ptr<Data> shptr)
{
    dataptr = shptr;
    this->initializeParticles();
}

void Summator::initializeParticles()
{
    // const unsigned int s = const unsigned int(dataptr->size);
    Particle particles[5];
    for (int i = 0; i < 5; i++)
    {
        particles[i].x = 0;
        particles[i].y = 0;
        particles[i].z = 0;
        particles[i].vx = 0;
        particles[i].vy = 0;
        particles[i].vz = 0;
    }

    dataptr->copyFromHostToDevice(particles);
}

hipError_t Summator::propagateParticles(Particle *particles, unsigned int size)
{
    propagateParticlesKernel<<<1, size>>>(dataptr->dev_parts);

    // Check for any errors launching the kernel
    dataptr->cudaStatus = hipGetLastError();
    if (dataptr->cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(dataptr->cudaStatus));
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
    dataptr->cudaStatus = hipDeviceSynchronize();
    if (dataptr->cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", dataptr->cudaStatus);
    }

    // Copy output vector from GPU buffer to host memory.
    dataptr->copyFromDeviceToHost(particles);

    return dataptr->cudaStatus;
}

hipError_t Summator::addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    // Copy input vectors from host memory to GPU buffers.
    dataptr->copyFromHostToDevice(a, dataptr->dev_a);
    dataptr->copyFromHostToDevice(b, dataptr->dev_b);

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dataptr->dev_c, dataptr->dev_a, dataptr->dev_b);

    // Check for any errors launching the kernel
    dataptr->cudaStatus = hipGetLastError();
    if (dataptr->cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(dataptr->cudaStatus));
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
    dataptr->cudaStatus = hipDeviceSynchronize();
    if (dataptr->cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", dataptr->cudaStatus);
    }

    // Copy output vector from GPU buffer to host memory.
    dataptr->copyFromDeviceToHost(dataptr->dev_c, c);

    return dataptr->cudaStatus;
}
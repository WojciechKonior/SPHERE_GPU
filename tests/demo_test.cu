#include <gtest/gtest.h>
#include <World.h>

TEST(dataClass, dataClassTest)
{
    const int arraySize = 5;
    const int a[arraySize] = {1, 2, 3, 4, 5};
    const int b[arraySize] = {10, 20, 30, 40, 50};
    int c[arraySize] = {0};

    std::shared_ptr<Data> dataptr(new Data(arraySize));
    std::shared_ptr<Summator> sum(new Summator(dataptr));
    hipError_t cudaStatus = sum->addWithCuda(c, a, b, arraySize);

    EXPECT_EQ(dataptr->cudaStatus, hipSuccess);
    for (int i = 0; i < arraySize; i++)
    {
        EXPECT_EQ(a[i] + b[i], c[i]);
    }
}

int main()
{
    testing::InitGoogleTest();
    return RUN_ALL_TESTS();
}